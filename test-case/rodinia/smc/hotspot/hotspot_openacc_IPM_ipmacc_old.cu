#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
using namespace std;
#define STR_SIZE  256


#define MAX_PD  (3.0e6)

#define PRECISION 0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100

#define FACTOR_CHIP 0.5


double t_chip = 0.0005;
double chip_height = 0.016;
double chip_width = 0.016;

double amb_temp = 80.0;
#define BLOCKSIZEX 16 
#define BLOCKSIZEY 16 
#define BLOCKSIZEXLOG 4
#define BLOCKSIZEYLOG 4


bool INRANGE(int rid, int cid, int iter, int innerIter, int tiler, int tilec, int blockdimr, int blockdimc)
{
    int localdimr = (blockdimr);
    int localdimc = (blockdimc);
    int localr = rid & (localdimr - 1);
    int localc = cid & (localdimc - 1);
    return ((localr > iter) && (localc > iter)) &&
        ((localr < (blockdimr - iter - 1)) && (localc < (blockdimc - iter - 1)));
}

__device__ bool __accelerator_INRANGE( int rid , int cid , int iter , int innerIter , int tiler , int tilec , int blockdimr , int blockdimc );
__global__ void __generated_kernel_region_0(double amb_temp,int dimrow,double* result,int row,int innerIter,double* power,int dimcol,double rRx,double stepCap,double delta,double* temp,double rRz,double rRy,int col);

void single_iteration(double *result, double *temp, double *power, int row, int col,
        double Cap, double Rx, double Ry, double Rz,
        double step, int innerIter, int *written)
{
    double delta;
    double stepCap = step / Cap;
    double rRx = 1 / Rx;
    double rRy = 1 / Ry;
    double rRz = 1 / Rz;
    int rs, cs;
#define TILEX (BLOCKSIZEX - 2 * innerIter)
#define TILEY (BLOCKSIZEY - 2 * innerIter)

    int dimrow = (row + ((2 * innerIter) * (row / TILEX + 1)));
    int dimcol = (col + ((2 * innerIter) * (col / TILEY + 1)));



    ipmacc_prompt((char*)"IPMACC: memory getting device pointer for temp\n");
    acc_present((void*)temp);
    ipmacc_prompt((char*)"IPMACC: memory getting device pointer for power\n");
    acc_present((void*)power);
    ipmacc_prompt((char*)"IPMACC: memory getting device pointer for result\n");
    acc_present((void*)result);
    ipmacc_prompt((char*)"IPMACC: memory getting device pointer for written\n");
    acc_present((void*)written);

    /* kernel call statement [0]*/
    {
        dim3 __ipmacc_gridDim(1,1,1);
        dim3 __ipmacc_blockDim(1,1,1);
        __ipmacc_blockDim.x=16;
        __ipmacc_gridDim.x=(((abs((int)((dimcol))-0))/(1))/__ipmacc_blockDim.x)+1;
        __ipmacc_blockDim.y=16;
        __ipmacc_gridDim.y=(((abs((int)((dimrow))-0))/(1))/__ipmacc_blockDim.y)+1;
        if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: (%u,%u,%u)\tblockDim: (%u,%u,%u)\n",__ipmacc_gridDim.x,__ipmacc_gridDim.y,__ipmacc_gridDim.z,__ipmacc_blockDim.x,__ipmacc_blockDim.y,__ipmacc_blockDim.z);
        __generated_kernel_region_0<<<__ipmacc_gridDim,__ipmacc_blockDim>>>(
                amb_temp,
                dimrow,
                (double*)acc_deviceptr((void*)result),
                row,
                innerIter,
                (double*)acc_deviceptr((void*)power),
                dimcol,
                rRx,
                stepCap,
                delta,
                (double*)acc_deviceptr((void*)temp),
                rRz,
                rRy,
                col);
    }
    /* kernel call statement*/
    if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
    {
        hipError_t err=hipDeviceSynchronize();
        if(err!=hipSuccess){
            printf("Kernel Launch Error! error code (%d)\n",err);
            assert(0&&"Launch Failure!\n");}
    }














}


void compute_tran_temp(double *result, int num_iterations, double *temp, double *power, int row, int col, int inner_iter, int *written)
{
#ifdef VERBOSE
    int i = 0;
#endif

    double grid_height = chip_height / row;
    double grid_width = chip_width / col;

    double Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
    double Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
    double Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
    double Rz = t_chip / (K_SI * grid_height * grid_width);

    double max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
    double step = PRECISION / max_slope;

#ifdef VERBOSE
    fprintf(stdout, "total iterations: %d s\tstep size: %g s\n", num_iterations, step);
    fprintf(stdout, "Rx: %g\tRy: %g\tRz: %g\tCap: %g\n", Rx, Ry, Rz, Cap);
#endif



    ipmacc_prompt((char*)"IPMACC: memory allocation temp\n");
    acc_create((void*)temp,(row*col+0)*sizeof(double));
    ipmacc_prompt((char*)"IPMACC: memory allocation written\n");
    acc_create((void*)written,(row*col+0)*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory allocation power\n");
    acc_create((void*)power,(row*col+0)*sizeof(double));
    ipmacc_prompt((char*)"IPMACC: memory allocation result\n");
    acc_create((void*)result,(row*col+0)*sizeof(double));
    ipmacc_prompt((char*)"IPMACC: memory copyin temp\n");
    acc_copyin((void*)temp,(row*col+0)*sizeof(double));
    ipmacc_prompt((char*)"IPMACC: memory copyin written\n");
    acc_copyin((void*)written,(row*col+0)*sizeof(int));
    ipmacc_prompt((char*)"IPMACC: memory copyin power\n");
    acc_copyin((void*)power,(row*col+0)*sizeof(double));


    {


        {

            for(int i = 0; i < num_iterations / inner_iter; i++)
            {
#ifdef VERBOSE
                fprintf(stdout, "iteration %d\n", i++);
#endif
                single_iteration(result, temp, power, row, col, Cap, Rx, Ry, Rz, step, inner_iter, written);
                double *tmp = temp;
                temp = result;
                result = tmp;
            }


        }
    }
    ipmacc_prompt((char*)"IPMACC: memory copyout temp\n");
    acc_copyout_and_keep((void*)temp,(row*col+0)*sizeof(double));
    ipmacc_prompt((char*)"IPMACC: memory copyout written\n");
    acc_copyout_and_keep((void*)written,(row*col+0)*sizeof(int));




#ifdef VERBOSE
    fprintf(stdout, "iteration %d\n", i++);
#endif
}

void fatal(char *s)
{
    fprintf(stderr, "error: %s\n", s);
    exit(1);
}

void read_input(double *vect, int grid_rows, int grid_cols, char *file)
{
    int i;
    FILE *fp;
    char str [STR_SIZE];
    double val;

    fp = fopen(file, "r");
    if (!fp) {
        fatal("file could not be opened for reading");
    }

    for (i = 0; i < grid_rows * grid_cols; i++) {
        fgets(str, STR_SIZE, fp);
        if (feof(fp)) {
            fatal("not enough lines in file");
        }
        if ((sscanf(str, "%lf", &val) != 1)) {
            fatal("invalid file format");
        }
        vect [i] = val;
    }

    fclose(fp);
}

void usage(int argc, char **argv)
{
    fprintf(stderr, "Usage: %s <grid_rows> <grid_cols> <inner_iter> <sim_time> <temp_file> <power_file>\n", argv [0]);
    fprintf(stderr, "\t<grid_rows>  - number of rows in the grid (positive integer)\n");
    fprintf(stderr, "\t<grid_cols>  - number of columns in the grid (positive integer)\n");
    fprintf(stderr, "\t<inner_iter>   - number of iterations within the region\n");
    fprintf(stderr, "\t<sim_time>   - number of iterations\n");
    fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
    fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
    exit(1);
}

int main(int argc, char **argv)
{
#ifdef __NVCUDA__
    acc_init(acc_device_nvcuda);
#endif
#ifdef __NVOPENCL__
    acc_init(acc_device_nvocl);

#endif

    int grid_rows, grid_cols, sim_time, inner_iter;
    double *temp, *power, *result;
    int *written;
    char *tfile, *pfile;


    if (argc != 7) {
        usage(argc, argv);
    }
    if ((grid_rows = atoi(argv [1])) <= 0 ||
            (grid_cols = atoi(argv [2])) <= 0 ||
            (sim_time = atoi(argv [4])) <= 0 ||
            (inner_iter = atoi(argv [3])) <= 0
       ) {
        usage(argc, argv);
    }


    temp = (double*)malloc(grid_rows * grid_cols * sizeof(double));
    power = (double*)malloc(grid_rows * grid_cols * sizeof(double));
    result = (double*)malloc(grid_rows * grid_cols * sizeof(double));
    written = (int*)malloc(grid_rows * grid_cols * sizeof(int));
    if (!temp || !power) {
        fatal("unable to allocate memory");
    }


    tfile = argv [5];
    pfile = argv [6];
    read_input(temp, grid_rows, grid_cols, tfile);
    read_input(power, grid_rows, grid_cols, pfile);
    int i;
    for (i = 0; i < grid_rows * grid_cols; i++) {
        written [i] = 0;
    }

    printf("Start computing the transient temperature\n");
    compute_tran_temp(result, sim_time, temp, power, grid_rows, grid_cols, inner_iter, written);
    printf("Ending simulation\n");

#ifdef VERBOSE
    fprintf(stdout, "Final Temperatures:\n");
#endif

#ifdef OUTPUT


    for (i = 0; i < grid_rows * grid_cols; i++) {
        fprintf(stdout, "%d\t%f\n", i, temp [i]);
    }
#endif

    free(temp);
    free(power);

    return 0;
}


__device__ bool __accelerator_INRANGE( int rid , int cid , int iter , int innerIter , int tiler , int tilec , int blockdimr , int blockdimc ) {
    int localdimr  = ( blockdimr ) ; int localdimc  = ( blockdimc ) ; int localr  = rid  & ( localdimr  - 1) ; int localc  = cid  & ( localdimc  - 1) ; return  (( localr  > iter ) && ( localc  > iter )) &&
        (( localr  < ( blockdimr  - iter  - 1)) && ( localc  < ( blockdimc  - iter  - 1))) ; 
}
__forceinline__ __device__ double __smc_select_0_temp(int index, double* g_array, double s_array[16+0+0][16+0+0], int startptr, int startptr2, int dim2size){
    // the pragmas are well-set. do not check the boundaries.
    int idx=index/dim2size;
    int idx2=index%dim2size;
    return s_array[idx-startptr][idx2-startptr2];
}
__forceinline__ __device__ double __smc_select_0_power(int index, double* g_array, double s_array[16+0+0][16+0+0], int startptr, int startptr2, int dim2size){
    // the pragmas are well-set. do not check the boundaries.
    int idx=index/dim2size;
    int idx2=index%dim2size;
    return s_array[idx-startptr][idx2-startptr2];
}

__device__ void __smc_write_0_temp(int index, double* g_array, double s_array[16+0+0][16+0+0], int startptr, int startptr2, int dim2size, double value){
    // the pragmas are well-set. do not check the boundaries.
    int idx=index/dim2size;
    int idx2=index%dim2size;
    s_array[idx-startptr][idx2-startptr2]=value;
}
__device__ void __smc_write_0_power(int index, double* g_array, double s_array[16+0+0][16+0+0], int startptr, int startptr2, int dim2size, double value){
    // the pragmas are well-set. do not check the boundaries.
    int idx=index/dim2size;
    int idx2=index%dim2size;
    s_array[idx-startptr][idx2-startptr2]=value;
}
__global__ void __generated_kernel_region_0(double amb_temp,int dimrow,double* result,int row,int innerIter,double* power,int dimcol,double rRx,double stepCap,double delta,double* temp,double rRz,double rRy,int col){
    int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
    int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
    int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
    int cs;
    int rs;

    /* declare the shared memory of temp */
    __shared__ double __kernel_smc_var_data_temp[16+0+0][16+0+0];
    /*__shared__*/ int __kernel_smc_startpointer_temp;
    /*__shared__*/ int __kernel_smc_endpointer_temp;
    /*__shared__*/ int __kernel_smc_startpointer_temp_2d;
    /*__shared__*/ int __kernel_smc_endpointer_temp_2d;
    __kernel_smc_endpointer_temp=-1;
    __kernel_smc_startpointer_temp=-1;
    __kernel_smc_endpointer_temp_2d=-1;
    __kernel_smc_startpointer_temp_2d=-1;
    /*{
      int iterator_of_smc=0;
      for(iterator_of_smc=threadIdx.x; iterator_of_smc<(16+0+0); iterator_of_smc+=blockDim.x){
    //__kernel_smc_var_data_temp[iterator_of_smc]=0;
    __kernel_smc_var_tag_temp[iterator_of_smc]=0;
    }
    __syncthreads();
    }*/

    /* declare the shared memory of power */
    __shared__ double __kernel_smc_var_data_power[16+0+0][16+0+0];
    /*__shared__*/ int __kernel_smc_startpointer_power;
    /*__shared__*/ int __kernel_smc_endpointer_power;
    /*__shared__*/ int __kernel_smc_startpointer_power_2d;
    /*__shared__*/ int __kernel_smc_endpointer_power_2d;
    __kernel_smc_endpointer_power=-1;
    __kernel_smc_startpointer_power=-1;
    __kernel_smc_endpointer_power_2d=-1;
    __kernel_smc_startpointer_power_2d=-1;
    /*{
      int iterator_of_smc=0;
      for(iterator_of_smc=threadIdx.x; iterator_of_smc<(16+0+0); iterator_of_smc+=blockDim.x){
    //__kernel_smc_var_data_power[iterator_of_smc]=0;
    __kernel_smc_var_tag_power[iterator_of_smc]=0;
    }
    __syncthreads();
    }*/
    {
        {
            {
                rs=0+(__kernel_getuid_y);
                if( rs < dimrow)
                {
                    int r = rs - ((rs >> BLOCKSIZEXLOG) * 2 * innerIter + innerIter);
                    {
                        cs=0+(__kernel_getuid_x);
                        if( cs < dimcol)
                        {
                            int c = cs - ((cs >> BLOCKSIZEYLOG) * 2 * innerIter + innerIter);
                            double new_temp;
                            bool compute = false;

                            {
                                int S = (r == (row - 1)) ? row - 1 : r + 1;
                                int N = (r == (0))    ? 0 : r - 1;
                                int W = (c == (0))    ? 0 : c - 1;
                                int E = (c == (col - 1)) ? col - 1 : c + 1;
                                //go on with the clause (temp[0:row:0:col:FETCH_CHANNEL:r:0:0:c:0:0:false:0:0:0:0],power[0:row:0:col:FETCH_CHANNEL:r:0:0:c:0:0:false:0:0:0:0])
                                { // fetch begins

                                    // FINDING TILE START
                                    __kernel_smc_startpointer_temp=r-0-threadIdx.y;
                                    __kernel_smc_startpointer_temp_2d=c-0-threadIdx.x;

                                    // FINDING DONE

                                    // FINDING TILE END
                                    bool lastcol=blockIdx.x==(gridDim.x-1);
                                    bool lastrow=blockIdx.y==(gridDim.y-1);
                                    __kernel_smc_endpointer_temp=(lastrow)?row-1:blockDim.y+__kernel_smc_startpointer_temp+0-1;
                                    __kernel_smc_endpointer_temp_2d=(lastcol)?col-1:blockDim.x+__kernel_smc_startpointer_temp_2d+0-1;
                                    // FINDING DONE
                                    //__fusion_merge_boundary_0()
                                    __kernel_smc_endpointer_power=     __kernel_smc_endpointer_temp;
                                    __kernel_smc_endpointer_power_2d=  __kernel_smc_endpointer_temp_2d;
                                    __kernel_smc_startpointer_power=   __kernel_smc_startpointer_temp;
                                    __kernel_smc_startpointer_power_2d=__kernel_smc_startpointer_temp_2d;

                                    int __ipmacc_length=__kernel_smc_endpointer_temp-__kernel_smc_startpointer_temp+1;
                                    int __ipmacc_length_2d=__kernel_smc_endpointer_temp_2d-__kernel_smc_startpointer_temp_2d+1;
                                    int kk=0,kk2=0;
                                    kk2=threadIdx.x;
                                    {
                                        int idx2=__kernel_smc_startpointer_temp_2d+kk2;
                                        if(idx2<(col) && idx2>=(0))
                                        {
                                            kk=threadIdx.y;
                                            {
                                                int idx=__kernel_smc_startpointer_temp+kk;
                                                if(idx<(row) && idx>=(0))
                                                {
                                                    __kernel_smc_var_data_temp[kk][kk2]=temp[idx*col+idx2];
                                                    //__kernel_smc_var_tag_temp[kk][kk2]=1;
                                                    //__fusion_merge_fetch_0()
                                                    __kernel_smc_var_data_power[kk][kk2]=power[idx*col+idx2];

                                                }
                                            }
                                        }
                                    }
                                    __syncthreads();
                                } // end of fetch
#define temp(index) __smc_select_0_temp(index, temp, __kernel_smc_var_data_temp, __kernel_smc_startpointer_temp, __kernel_smc_startpointer_temp_2d, col)

                                // 5 unique indexes
                                // [0] S*col+c
#define __ipmacc_smc_index_temp_0_dim1 S-__kernel_smc_startpointer_temp
#define __ipmacc_smc_index_temp_0_dim2 c-__kernel_smc_startpointer_temp_2d
                                // [1] N*col+c
#define __ipmacc_smc_index_temp_1_dim1 N-__kernel_smc_startpointer_temp
#define __ipmacc_smc_index_temp_1_dim2 c-__kernel_smc_startpointer_temp_2d
                                // [2] r*col+c
#define __ipmacc_smc_index_temp_2_dim1 r-__kernel_smc_startpointer_temp
#define __ipmacc_smc_index_temp_2_dim2 c-__kernel_smc_startpointer_temp_2d
                                // [3] r*col+E
#define __ipmacc_smc_index_temp_3_dim1 r-__kernel_smc_startpointer_temp
#define __ipmacc_smc_index_temp_3_dim2 E-__kernel_smc_startpointer_temp_2d
                                // [4] r*col+W
#define __ipmacc_smc_index_temp_4_dim1 r-__kernel_smc_startpointer_temp
#define __ipmacc_smc_index_temp_4_dim2 W-__kernel_smc_startpointer_temp_2d
                                { // fetch begins
                                } // end of fetch
#define power(index) __smc_select_0_power(index, power, __kernel_smc_var_data_power, __kernel_smc_startpointer_power, __kernel_smc_startpointer_power_2d, col)

                                // 1 unique indexes
                                // [0] r*col+c
#define __ipmacc_smc_index_power_0_dim1 r-__kernel_smc_startpointer_power
#define __ipmacc_smc_index_power_0_dim2 c-__kernel_smc_startpointer_power_2d

                                {


                                    {
                                        for(int iter = 0; iter < innerIter; iter++)
                                        {
                                            compute = false;
                                            if ((r >= 0) && (c >= 0) && (r < row) && (c < col) &&
                                                    __accelerator_INRANGE(rs, cs, iter, innerIter, TILEX, TILEY, BLOCKSIZEX, BLOCKSIZEY)) {

                                                compute = true;
                                                delta = (stepCap) * (__kernel_smc_var_data_power[__ipmacc_smc_index_power_0_dim1][__ipmacc_smc_index_power_0_dim2] /* replacing power [r * col + c]*/  +
                                                        (__kernel_smc_var_data_temp[__ipmacc_smc_index_temp_0_dim1][__ipmacc_smc_index_temp_0_dim2] /* replacing temp [S * col + c]*/  + __kernel_smc_var_data_temp[__ipmacc_smc_index_temp_1_dim1][__ipmacc_smc_index_temp_1_dim2] /* replacing temp [N * col + c]*/  - 2.0 * __kernel_smc_var_data_temp[__ipmacc_smc_index_temp_2_dim1][__ipmacc_smc_index_temp_2_dim2] /* replacing temp [r * col + c]*/ ) * rRy +
                                                        (__kernel_smc_var_data_temp[__ipmacc_smc_index_temp_3_dim1][__ipmacc_smc_index_temp_3_dim2] /* replacing temp [r * col + E]*/  + __kernel_smc_var_data_temp[__ipmacc_smc_index_temp_4_dim1][__ipmacc_smc_index_temp_4_dim2] /* replacing temp [r * col + W]*/  - 2.0 * __kernel_smc_var_data_temp[__ipmacc_smc_index_temp_2_dim1][__ipmacc_smc_index_temp_2_dim2] /* replacing temp [r * col + c]*/ ) * rRx +
                                                        (amb_temp - __kernel_smc_var_data_temp[__ipmacc_smc_index_temp_2_dim1][__ipmacc_smc_index_temp_2_dim2] /* replacing temp [r * col + c]*/ ) * rRz);
                                                new_temp = delta + __kernel_smc_var_data_temp[__ipmacc_smc_index_temp_2_dim1][__ipmacc_smc_index_temp_2_dim2] /* replacing temp [r * col + c]*/ ;
                                            }
                                            if (iter == (innerIter - 1)) {
                                                break;
                                            }
                                            if (compute) {

                                                __syncthreads();
                                                __kernel_smc_var_data_temp[__ipmacc_smc_index_temp_2_dim1][__ipmacc_smc_index_temp_2_dim2]= new_temp;
                                                __syncthreads();
                                            }
                                        }
                                    }
                                }
#undef temp
#undef power

                                //end up with the clause (temp[0:row:0:col:FETCH_CHANNEL:r:0:0:c:0:0:false:0:0:0:0],power[0:row:0:col:FETCH_CHANNEL:r:0:0:c:0:0:false:0:0:0:0])
                            }



                            if (compute) {
                                result [r * col + c] = new_temp;   

                            }
                        }

                    }
                }

            }
        }
    }
}

