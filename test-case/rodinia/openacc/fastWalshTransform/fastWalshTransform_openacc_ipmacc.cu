#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#ifdef __cplusplus
#include "openacc_container.h"
#endif

#include <hip/hip_runtime.h>





extern int dataN;
extern int kernelN;

  __global__ void __generated_kernel_region_0(float * h_Kernel,float * h_Data,float * h_Result,const int  kernelNL,const int  dataNL);
 
void dyadicConvolutionCPU_openacc(
  float *h_Result_f,
  float *h_Data_f,
  float *h_Kernel_f,
  int log2dataN,
  int log2kernelN)
{
  
  
  const int dataNL = 1 << log2dataN;
  const int kernelNL = 1 << log2kernelN;

  float *h_Result = (float*)h_Result_f;
  float *h_Data = (float*)h_Data_f;
  float *h_Kernel = (float*)h_Kernel_f;

    

	ipmacc_prompt((char*)"IPMACC: memory allocation h_Result\n");
acc_present_or_create((void*)h_Result,(dataN+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory allocation h_Data\n");
acc_present_or_create((void*)h_Data,(dataN+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory allocation h_Kernel\n");
acc_present_or_create((void*)h_Kernel,(kernelN+0)*sizeof(float ));
	ipmacc_prompt((char*)"IPMACC: memory copyin h_Data\n");
acc_pcopyin((void*)h_Data,(dataN+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory copyin h_Kernel\n");
acc_pcopyin((void*)h_Kernel,(kernelN+0)*sizeof(float ));

/* kernel call statement [0]*/
{
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",(((abs((int)((dataNL))-0))/(1)))/256+(((((abs((int)((dataNL))-0))/(1)))%(256))==0?0:1),256);
__generated_kernel_region_0<<<(((abs((int)((dataNL))-0))/(1)))/256+(((((abs((int)((dataNL))-0))/(1)))%(256))==0?0:1),256>>>(
(float *)acc_deviceptr((void*)h_Kernel),
(float *)acc_deviceptr((void*)h_Data),
(float *)acc_deviceptr((void*)h_Result),
kernelNL,
dataNL);
}
/* kernel call statement*/
	ipmacc_prompt((char*)"IPMACC: memory copyout h_Result\n");
acc_copyout_and_keep((void*)h_Result,(dataN+0)*sizeof(float ));
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
{
hipError_t err=hipDeviceSynchronize();
if(err!=hipSuccess){
printf("Kernel Launch Error! error code (%d)\n",err);
assert(0&&"Launch Failure!\n");}
}



}




  __global__ void __generated_kernel_region_1(int  stride,int  base,float * h_Output);
 
  __global__ void __generated_kernel_region_2(const int  N,int  stride,int  base,float * h_Output);
 
void fwtCPU_openacc(float *h_Output_f, float *h_Input_f, int log2N)
{
  const int N = 1 << log2N;
  float *h_Output = (float*)h_Output_f;
  float *h_Input = (float*)h_Input_f;

  for (int pos = 0; pos < N; pos++) {
    h_Output [pos] = h_Input [pos];
  }

  int stride, baseI, base;
  
    

	ipmacc_prompt((char*)"IPMACC: memory allocation h_Output\n");
acc_present_or_create((void*)h_Output,(dataN+0)*sizeof(float ));
ipmacc_prompt((char*)"IPMACC: memory allocation h_Input\n");
acc_present_or_create((void*)h_Input,(dataN+0)*sizeof(float ));
	ipmacc_prompt((char*)"IPMACC: memory copyin h_Input\n");
acc_pcopyin((void*)h_Input,(dataN+0)*sizeof(float ));


{


  
		for(stride = N / 2; stride >= 1; stride >>= 1)
 {
    printf("calling %d\n", stride);
    
    if (stride > 128) {
      
			for(baseI = 0; baseI < (N / (2 * stride) + 1); baseI++)
 {
        
        base = baseI * 2 * stride;
        if (base < N) {
                    

								ipmacc_prompt((char*)"IPMACC: memory getting device pointer for h_Output\n");
acc_present((void*)h_Output);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for h_Input\n");
acc_present((void*)h_Input);

/* kernel call statement [1, 2]*/
{
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 1 > gridDim: %d\tblockDim: %d\n",(((abs((int)((stride))-0))/(1)))/256+(((((abs((int)((stride))-0))/(1)))%(256))==0?0:1),256);
__generated_kernel_region_1<<<(((abs((int)((stride))-0))/(1)))/256+(((((abs((int)((stride))-0))/(1)))%(256))==0?0:1),256>>>(
stride,
base,
(float *)acc_deviceptr((void*)h_Output));
}
/* kernel call statement*/
				if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
{
hipError_t err=hipDeviceSynchronize();
if(err!=hipSuccess){
printf("Kernel Launch Error! error code (%d)\n",err);
assert(0&&"Launch Failure!\n");}
}



        }
      }


    }else{
            

						ipmacc_prompt((char*)"IPMACC: memory getting device pointer for h_Output\n");
acc_present((void*)h_Output);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for h_Input\n");
acc_present((void*)h_Input);

/* kernel call statement [1, 3]*/
{
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 2 > gridDim: %d\tblockDim: %d\n",(((abs((int)((((N/(2*stride))+1)))-0))/(1)))/256+(((((abs((int)((((N/(2*stride))+1)))-0))/(1)))%(256))==0?0:1),256);
__generated_kernel_region_2<<<(((abs((int)((((N/(2*stride))+1)))-0))/(1)))/256+(((((abs((int)((((N/(2*stride))+1)))-0))/(1)))%(256))==0?0:1),256>>>(
N,
stride,
base,
(float *)acc_deviceptr((void*)h_Output));
}
/* kernel call statement*/
			if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
{
hipError_t err=hipDeviceSynchronize();
if(err!=hipSuccess){
printf("Kernel Launch Error! error code (%d)\n",err);
assert(0&&"Launch Failure!\n");}
}



    }
  }


}
	ipmacc_prompt((char*)"IPMACC: memory copyout h_Output\n");
acc_copyout_and_keep((void*)h_Output,(dataN+0)*sizeof(float ));



}



 __global__ void __generated_kernel_region_0(float * h_Kernel,float * h_Data,float * h_Result,const int  kernelNL,const int  dataNL){
int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
{
{
{
int i=0+(__kernel_getuid_x);
if( i < dataNL)
{
    double sum = 0;
for(int j = 0; j < kernelNL; j++)
{
      sum += h_Data [i ^ j] * h_Kernel [j];
    }
h_Result [i] = (float)sum;
  }

}
}
}
}

 __global__ void __generated_kernel_region_1(int  stride,int  base,float * h_Output){
int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
{
{
{
int j=0+(__kernel_getuid_x);
if( j < stride)
{
            
            int i0 = base + j + 0;
            int i1 = base + j + stride;

            float T1 = h_Output [i0];
            float T2 = h_Output [i1];
            h_Output [i0] = T1 + T2;
            h_Output [i1] = T1 - T2;
          }

}
}
}
}

 __global__ void __generated_kernel_region_2(const int  N,int  stride,int  base,float * h_Output){
int __kernel_getuid_x=threadIdx.x+blockIdx.x*blockDim.x;
int __kernel_getuid_y=threadIdx.y+blockIdx.y*blockDim.y;
int __kernel_getuid_z=threadIdx.z+blockIdx.z*blockDim.z;
int  baseI;
{
{
{
 baseI=0+(__kernel_getuid_x);
if( baseI < (N / (2 * stride) + 1))
{
        
        base = baseI * 2 * stride;
        if (base < N) {
for(int j = 0; j < stride; j++)
{
            
            int i0 = base + j + 0;
            int i1 = base + j + stride;

            float T1 = h_Output [i0];
            float T2 = h_Output [i1];
            h_Output [i0] = T1 + T2;
            h_Output [i1] = T1 - T2;
          }
}
      }

}
}
}
}

