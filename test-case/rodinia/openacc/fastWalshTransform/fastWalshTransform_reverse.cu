#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



#ifndef FWT_KERNEL_CUH
#define FWT_KERNEL_CUH
#ifndef fwt_kernel_cuh
#define fwt_kernel_cuh



///////////////////////////////////////////////////////////////////////////////
// Elementary(for vectors less than elementary size) in-shared memory 
// combined radix-2 + radix-4 Fast Walsh Transform 
///////////////////////////////////////////////////////////////////////////////
#define ELEMENTARY_LOG2SIZE 11

void fwtBatch1Kernel(float *d_Output, float *d_Input, int log2N, int DIMX){
    for(tidx=0; tidx<DIMX; tidx++){
        const int    N = 1 << log2N;
        const int base = (tidx/BLOCKDIMX)<<log2N; //blockIdx.x << log2N;

        //(2 ** 11) * 4 bytes == 8KB -- maximum s_data[] size for G80
        float s_data[N];
        float *d_Src = d_Input  + base;
        float *d_Dst = d_Output + base;

        for(int pos = threadIdx.x; pos < N; pos += blockDim.x)
            s_data[pos] = d_Input[base+pos];//d_Src[pos];

        //Main radix-4 stages
        const int pos = tidx%BLOCKDIMY;//threadIdx.x;
        for(int stride = base + (N >> 2); stride > (base+0); stride >>= 2){
            int lo =        (pos & (stride - 1));
            int i0 = base + (((pos - lo) << 2) + lo);
            int i1 = base + (i0 + stride);
            int i2 = base + (i1 + stride);
            int i3 = base + (i2 + stride);

            __syncthreads();
            float D0 = d_Input[i0];//s_data[i0];
            float D1 = d_Input[i1];//s_data[i1];
            float D2 = d_Input[i2];//s_data[i2];
            float D3 = d_Input[i3];//s_data[i3];

            float T;
            T = D0; D0         = D0 + D2; D2         = T - D2;
            T = D1; D1         = D1 + D3; D3         = T - D3;
            T = D0; d_Input[i0]/*s_data[i0]*/ = D0 + D1; d_Input[i1]/*s_data[i1]*/ = T - D1;
            T = D2; d_Input[i2]/*s_data[i2]*/ = D2 + D3; d_Input[i3]/*s_data[i3]*/ = T - D3;
        }

        //Do single radix-2 stage for odd power of two
        if(log2N & 1){
            __syncthreads();
            for(int pos = threadIdx.x; pos < N / 2; pos += blockDim.x){
                int i0 = pos << 1;
                int i1 = i0 + 1;

                float D0 = s_data[i0];
                float D1 = s_data[i1];
                s_data[i0] = D0 + D1;
                s_data[i1] = D0 - D1;
            }
        }

        __syncthreads();
        for(int pos = threadIdx.x; pos < N; pos += blockDim.x)
            d_Dst[pos] = s_data[pos];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Single in-global memory radix-4 Fast Walsh Transform pass
// (for strides exceeding elementary vector size)
////////////////////////////////////////////////////////////////////////////////
void fwtBatch2Kernel(
    float *d_Output,
    float *d_Input,
    int stride,
    int DIMX,
    int DIMY
){
    //#define DIMX (blockDim.x*gridDim.x)
    //#define DIMY (blockDim.y*gridDim.y)
    //#define BLOCKDIMY 16 
    for(tidy=0; tidy<DIMY; tidy++){
        for(tidx=0; tidx<DIMX; tidx++){
            const int pos = tidx; // blockIdx.x * blockDim.x + threadIdx.x;
            const int   N = DIMX*4;//blockDim.x *  gridDim.x * 4;

            //float *d_Src = d_Input + blockIdx.y * N;
            //float *d_Dst = d_Output + blockIdx.y * N;
            float offset = tidy*N;

            int lo = pos & (stride - 1);
            int i0 = ((pos - lo) << 2) + lo;
            int i1 = i0 + stride;
            int i2 = i1 + stride;
            int i3 = i2 + stride;

            float D0 = d_Input[offset+i0];
            float D1 = d_Input[offset+i1];
            float D2 = d_Input[offset+i2];
            float D3 = d_Input[offset+i3];

            float T;
            T = D0; D0        = D0 + D2;           D2 = T - D2;
            T = D1; D1        = D1 + D3;           D3 = T - D3;
            T = D0; d_Output[offset+i0] = D0 + D1; d_Output[offset+i1] = T - D1;
            T = D2; d_Output[offset+i2] = D2 + D3; d_Output[offset+i3] = T - D3;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Put everything together: batched Fast Walsh Transform CPU front-end
////////////////////////////////////////////////////////////////////////////////
void fwtBatchGPU(float *d_Data, int M, int log2N){
    const int THREAD_N = 256;

    int N = 1 << log2N;
    //dim3 grid((1 << log2N) / (4 * THREAD_N), M, 1);
    for(; log2N > ELEMENTARY_LOG2SIZE; log2N -= 2, N >>= 2, M <<= 2){
        //fwtBatch2Kernel<<<grid, THREAD_N>>>(d_Data,
        fwtBatch2Kernel(d_Data,
            d_Data,
            N / 4,
            (1 << log2N) / (4 * THREAD_N),
            M);
        //cutilCheckMsg("fwtBatch2Kernel() execution failed\n");
    }

    fwtBatch1Kernel<<<M, N / 4, N * sizeof(float)>>>(
        d_Data,
        d_Data,
        log2N
    );
    cutilCheckMsg("fwtBatch1Kernel() execution failed\n");
}



////////////////////////////////////////////////////////////////////////////////
// Modulate two arrays
////////////////////////////////////////////////////////////////////////////////
__global__ void modulateKernel(float *d_A, float *d_B, int N){
    int        tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    float     rcpN = 1.0f / (float)N;

    for(int pos = tid; pos < N; pos += numThreads)
        d_A[pos] *= d_B[pos] * rcpN;
}

//Interface to modulateKernel()
void modulateGPU(float *d_A, float *d_B, int N){
    modulateKernel<<<128, 256>>>(d_A, d_B, N);
}



#endif
#endif
