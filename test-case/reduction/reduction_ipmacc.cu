#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <iostream>
#include "openacc.h"

using namespace std;


  __global__ void __generated_kernel_region_0(int* arr,int arr_size,int* sum__ipmacc_reductionarray_internal);
 
int main()
{
  int arr [100000];
  int sum = 0;
  int sum2 = 0;
  int arr_size = 1;

    #ifdef __NVCUDA__
  acc_init(acc_device_nvcuda);
    #endif
    #ifdef __NVOPENCL__
  acc_init(acc_device_nvocl);
  
    #endif



  srand(time(NULL));

  while (arr_size < 100000) {
    sum = 0;
    sum2 = 0;
    for (int i = 0; i < arr_size; ++i) {
      arr [i] = rand() % 100;
    }
    
    
    

	ipmacc_prompt((char*)"IPMACC: memory allocation arr\n");
acc_create((void*)arr,100000*sizeof(int));
	ipmacc_prompt((char*)"IPMACC: memory copyin arr\n");
acc_copyin((void*)arr,100000*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation sum\n");
int* __ipmacc_reduction_array_sum=NULL;
if(__ipmacc_reduction_array_sum==NULL){
__ipmacc_reduction_array_sum=(int*)malloc((((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));
acc_create((void*)__ipmacc_reduction_array_sum,(((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));
for(int __ipmacc_initialize_rv=0; __ipmacc_initialize_rv<(((abs((int)((arr_size))-0))/(1))/256+1); __ipmacc_initialize_rv++){
__ipmacc_reduction_array_sum[__ipmacc_initialize_rv]= 0;
}
acc_pcopyin((void*)__ipmacc_reduction_array_sum,(((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));
}

/* kernel call statement [0]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",(((abs((int)((arr_size))-0))/(1)))/256+1,256);
__generated_kernel_region_0<<<(((abs((int)((arr_size))-0))/(1)))/256+1,256>>>(
(int*)acc_deviceptr((void*)arr),
arr_size,
(int*)acc_deviceptr((void*)__ipmacc_reduction_array_sum));
/* kernel call statement*/
	ipmacc_prompt((char*)"IPMACC: memory copyout arr\n");
acc_copyout_and_keep((void*)arr,100000*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory copyout sum\n");
acc_copyout_and_keep((void*)__ipmacc_reduction_array_sum,(((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));

/* second-level reduction on sum */
{
int __kernel_reduction_iterator=0;
{
int bound = (((abs((int)((arr_size))-0))/(1))/256+1)-1;
for(__kernel_reduction_iterator=bound; __kernel_reduction_iterator>0; __kernel_reduction_iterator-=1){
__ipmacc_reduction_array_sum[__kernel_reduction_iterator-1]=__ipmacc_reduction_array_sum[__kernel_reduction_iterator-1]+__ipmacc_reduction_array_sum[__kernel_reduction_iterator];
}
}
}
sum=__ipmacc_reduction_array_sum[0];
free(__ipmacc_reduction_array_sum);
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();



    for (int i = 0; i < arr_size; ++i) {
      sum2 += arr [i];
    }
    cout << "cpu result:" << sum2 << endl;
    cout << "gpu result:" << sum << endl;
    cout << "array size:" << arr_size << endl;
    arr_size++;
    assert(sum == sum2);
  }

  return 0;
}


 __global__ void __generated_kernel_region_0(int* arr,int arr_size,int* sum__ipmacc_reductionarray_internal){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
__shared__ int __kernel_reduction_shmem_int[256];
int __kernel_reduction_iterator=0;
int sum;{
{
{
int i=0+(__kernel_getuid);
if( i < arr_size)
{ //opened for private and reduction
/*private:+:sum*/
{ //start of reduction region for sum 
int sum=0;

{
      sum += arr [i];
    }
/*reduction:+:sum*/

/* reduction on sum */
__syncthreads();
__kernel_reduction_shmem_int[threadIdx.x]=sum;
__syncthreads();
for(__kernel_reduction_iterator=blockDim.x/2;__kernel_reduction_iterator>0; __kernel_reduction_iterator>>=1) {
if(threadIdx.x<__kernel_reduction_iterator){
__kernel_reduction_shmem_int[threadIdx.x]=__kernel_reduction_shmem_int[threadIdx.x]+__kernel_reduction_shmem_int[threadIdx.x+__kernel_reduction_iterator];
}
__syncthreads();
}
}// the end of sum scope
if(threadIdx.x==0){
sum__ipmacc_reductionarray_internal[blockIdx.x]=__kernel_reduction_shmem_int[0];
}

} // closed for reduction-end

}
}
}
}

