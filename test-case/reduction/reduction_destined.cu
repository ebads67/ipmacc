#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <iostream>

using namespace std;


__global__ void __generated_kernel_region_0(int* arr,int arr_size,int* sum__ipmacc_reductionarray_internal);

int main()
{
#define SIZE 100000
	int arr [SIZE];
	int sum = 0;
	int sum2 = 0;
	int arr_size = 5400;

	srand(time(NULL));

	while (arr_size < 10000) {
		sum = 0;
		sum2 = 0;
		for (int i = 0; i < arr_size; ++i) {
			arr [i] = rand() % 100;
		}




		ipmacc_prompt((char*)"IPMACC: memory allocation arr\n");
		acc_create((void*)arr,100000*sizeof(int));
		ipmacc_prompt((char*)"IPMACC: memory copyin arr\n");
		acc_copyin((void*)arr,100000*sizeof(int));
		ipmacc_prompt((char*)"IPMACC: memory allocation sum\n");
		int* __ipmacc_reduction_array_sum=NULL;
		int dimGrid=(((abs((int)((arr_size))-0))/(1)))/256+1;
		if(__ipmacc_reduction_array_sum==NULL){
			__ipmacc_reduction_array_sum=(int*)malloc(dimGrid*sizeof(int));
			//__ipmacc_reduction_array_sum=(int*)malloc((((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));
			//for(int kk=0; kk<(((abs((int)((arr_size))-0))/(1))/256+1); kk++){
			//	__ipmacc_reduction_array_sum[kk]=-1;
			//}
			acc_create((void*)__ipmacc_reduction_array_sum,(((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));
			//acc_copyin((void*)__ipmacc_reduction_array_sum,(((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));
		}

		/* kernel call statement [0]*/
		//cout << "array size:" << arr_size << endl;
		assert((int*)acc_deviceptr((void*)arr));
		assert((int*)acc_deviceptr((void*)__ipmacc_reduction_array_sum));
		int dimBlock=256;
		//printf("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",dimGrid,dimBlock);
		__generated_kernel_region_0<<<dimGrid,dimBlock>>>(
				(int*)acc_deviceptr((void*)arr),
				arr_size,
				(int*)acc_deviceptr((void*)__ipmacc_reduction_array_sum));
		hipDeviceSynchronize();
		//cout<<"Error> "<<hipGetErrorString(hipPeekAtLastError())<<endl;
		/* kernel call statement*/
		ipmacc_prompt((char*)"IPMACC: memory copyout arr\n");
		acc_copyout_and_keep((void*)arr,100000*sizeof(int));
		ipmacc_prompt((char*)"IPMACC: memory copyout sum\n");
		acc_copyout_and_keep((void*)__ipmacc_reduction_array_sum,(((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));

		/* second-level reduction on sum */
		{
			for(int kk=0; kk<dimGrid; kk++){
				printf("\t%d\n",__ipmacc_reduction_array_sum[kk]);
			}
			acc_create((void*)__ipmacc_reduction_array_sum,(((abs((int)((arr_size))-0))/(1))/256+1)*sizeof(int));
			int __kernel_reduction_iterator=0;
			int count= (((abs((int)((arr_size))-0))/(1))%256) == 0? (((abs((int)((arr_size))-0))/(1))/256+1)-2: (((abs((int)((arr_size))-0))/(1))/256+1)-1;
			for(__kernel_reduction_iterator=count; __kernel_reduction_iterator>0; __kernel_reduction_iterator-=1){
			//for(__kernel_reduction_iterator=(((abs((int)((arr_size))-0))/(1))/256+1)-1; __kernel_reduction_iterator>0; __kernel_reduction_iterator-=1){
				__ipmacc_reduction_array_sum[__kernel_reduction_iterator-1]=__ipmacc_reduction_array_sum[__kernel_reduction_iterator-1]+__ipmacc_reduction_array_sum[__kernel_reduction_iterator];
			}
		}
		sum=__ipmacc_reduction_array_sum[0];
		if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
		hipDeviceSynchronize();



		for (int i = 0; i < arr_size; ++i) {
			sum2 += arr [i];
		}
		cout << "cpu result:" << sum2 << endl;
		cout << "gpu result:" << sum << endl;
		cout << "array size:" << arr_size << endl;
		cout << "===================" << endl;
		arr_size++;
		assert(sum == sum2);
	}

	return 0;
}


__global__ void __generated_kernel_region_0(int* arr,int arr_size,int* sum__ipmacc_reductionarray_internal){
	int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
	__shared__ int __kernel_reduction_shmem_int[256];
	__kernel_reduction_shmem_int[threadIdx.x]=0;
	int __kernel_reduction_iterator=0;
	int sum;
	{
		{
			{
				int i=0+(__kernel_getuid);
				if( i < arr_size)
				{ //opened for private and reduction
					/*private:+:sum*/
					{ //start of reduction region for sum 
						int sum=0;

						{
							sum += arr [i];
						}
						/*reduction:+:sum*/

						/* reduction on sum */
						__syncthreads();
						__kernel_reduction_shmem_int[threadIdx.x]=sum;//threadIdx.x;//;
						__syncthreads();
						
					//	   for(__kernel_reduction_iterator=256; __kernel_reduction_iterator>1; __kernel_reduction_iterator=__kernel_reduction_iterator/2){
					//	   if(threadIdx.x<__kernel_reduction_iterator && threadIdx.x>=__kernel_reduction_iterator/2){
					//	   __kernel_reduction_shmem_int[threadIdx.x-(__kernel_reduction_iterator/2)]=__kernel_reduction_shmem_int[threadIdx.x-(__kernel_reduction_iterator/2)]+__kernel_reduction_shmem_int[threadIdx.x];
					//	   }
					//	   __syncthreads();
					//	   }
						 
						for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
							if(threadIdx.x<s)
							{
								__kernel_reduction_shmem_int[threadIdx.x] += __kernel_reduction_shmem_int[threadIdx.x + s];
							}
							__syncthreads();
						}
					}// the end of sum scope
					if(threadIdx.x==0){
						//int tot_sum=0;
						//for (int s=0; s<(int)blockDim.x; s++) {
						//	tot_sum+=arr[s];//__kernel_reduction_shmem_int[s];
						//}
						//sum__ipmacc_reductionarray_internal[blockIdx.x]=tot_sum;//threadIdx.x+1;
						sum__ipmacc_reductionarray_internal[blockIdx.x]=__kernel_reduction_shmem_int[0];
					}

				} // closed for reduction-end

			}
		}
	}
}

