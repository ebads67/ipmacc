#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <openacc.h>
#define IPMACC_MAX1(A)   (A)
#define IPMACC_MAX2(A,B) (A>B?A:B)
#define IPMACC_MAX3(A,B,C) (A>B?(A>C?A:(B>C?B:C)):(B>C?C:B))
#include <hip/hip_runtime.h>






















#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define __STDC_LIMIT_MACROS
#include <stdint.h> 

#include <climits>
#include "arraybench.h"
#include <sys/time.h>

double btest [IDA];
double atest [IDA];
int nthreads, delaylength, innerreps;
double times [OUTERREPS + 1], reftime, refsd;

time_t starttime = 0;
timeval tim;



int main(int argv, char **argc)
{
#ifdef __NVCUDA__
  acc_init(acc_device_nvcuda);
#endif
#ifdef __NVOPENCL__
  acc_init(acc_device_nvocl);
  
#endif



  printf(" *******************************************************\n");

  delaylength = 500;
  innerreps = 100;
  
  

  
  

  
  

#ifdef OMPVER2
  
  
#endif

  
  copyintest();

  
  copyouttest();

  
  createtest();


  
  reductiontest();

  
  kerneltest();


  
  



  delaylength = 500;
  innerreps = 100;
}

static int firstcall = 1;


double get_time_of_day_()
{
  struct timeval ts;

  double t;

  int err;

  err = gettimeofday(&ts, NULL);

  t = (double)(ts.tv_sec - starttime) + (double)ts.tv_usec * 1.0e-6;

  return t;
}

void init_time_of_day_()
{
  struct  timeval ts;
  int err;

  err = gettimeofday(&ts, NULL);
  starttime = ts.tv_sec;
}

double getclock(void)
{
  double time;
  double get_time_of_day_(void);
  void init_time_of_day_(void);

  if (firstcall) {
    init_time_of_day_();
    firstcall = 0;
  }
  time = get_time_of_day_();
  return time;
}


void delay(int delaylength, double a [1])
{
  int i;
  a [0] = 1.0;
  for (i = 0; i < delaylength; i++) {
    a [0] += i;
  }
  
}

void refer()
{
  int j, k;
  int i = 0;
  double start;
  double meantime, sd, hm;
  double a [1];
  

  printf("\n");
  printf("--------------------------------------------------------\n");
  printf("Computing reference time 1\n");

  for (k = 0; k <= OUTERREPS; k++) {
    start = getclock();
    for (j = 0; j < innerreps; j++) {
      delay(delaylength, a);
      i++;
    }
    times [k] = (getclock() - start) * 1.0e6 / (double)innerreps;
  }

  stats(&meantime, &sd, &hm);

  
  printf("Reference_time_1 =                        %10.3f microseconds +/- %10.3f\n", hm, CONF95 * sd);

  reftime = meantime;
  refsd = sd;
}

void testfirstprivnew()
{
  int n, j, k;
  double start;
  double meantime, sd;
  

  n = IDA;
  printf("\n");
  printf("--------------------------------------------------------\n");
  printf("Computing FIRSTPRIVATE %d time\n", n);

  for (k = 0; k <= OUTERREPS; k++) {
    start = getclock();
    
    for (j = 0; j < innerreps; j++) {
      
      {
        delay(delaylength, atest);
      }
    }
    times [k] = (getclock() - start) * 1.0e6 / (double)innerreps;
  }

  

  
  printf("FIRSTPRIVATE time =                           %10.3f microseconds +/- %10.3f\n", meantime, CONF95 * sd);
  printf("FIRSTPRIVATE overhead =                       %10.3f microseconds +/- %10.3f\n", meantime - reftime, CONF95 * (sd + refsd));
}

void testprivnew()
{
  int n, j, k;
  double start;
  double meantime, sd;
  

  n = IDA;
  printf("\n");
  printf("--------------------------------------------------------\n");
  printf("Computing PRIVATE %d time\n", n);

  for (k = 0; k <= OUTERREPS; k++) {
    start = getclock();
    for (j = 0; j < innerreps; j++) {
      
      {
        delay(delaylength, atest);
      }
    }
    times [k] = (getclock() - start) * 1.0e6 / (double)innerreps;
  }

  

  
  printf("PRIVATE time =                           %10.3f microseconds +/- %10.3f\n", meantime, CONF95 * sd);
  printf("PRIVATE overhead =                       %10.3f microseconds +/- %10.3f\n", meantime - reftime, CONF95 * (sd + refsd));
}

#ifdef OMPVER2
void testcopyprivnew()
{
  int n, j, k;
  double start;
  double meantime, sd;
  
  n = IDA;
  printf("\n");
  printf("--------------------------------------------------------\n");
  printf("Computing COPYPRIVATE %d time\n", n);

  for (k = 0; k <= OUTERREPS; k++) {
    start = getclock();
    for (j = 0; j < innerreps; j++) {
      
      {
        delay(delaylength, btest);
      }
    }
    times [k] = (getclock() - start) * 1.0e6 / (double)innerreps;
  }

  

  printf("COPYPRIVATE time =                           %10.3f microseconds +/- %10.3f\n", meantime, CONF95 * sd);
  printf("COPYPRIVATE overhead =                       %10.3f microseconds +/- %10.3f\n", meantime - reftime, CONF95 * (sd + refsd));
}

#endif
void createtest()
{
  int n, j, k;
  double start, end;
  double meantime, sd, hm;
  
  n = IDA;
  
  
  


	ipmacc_prompt((char*)"IPMACC: memory allocation btest\n");
acc_create((void*)btest,IDA*sizeof(double));
	

{


  
		for(k = 0; k <= OUTERREPS; k++)
 {
    
    gettimeofday(&tim, NULL);
    start = tim.tv_sec * 1000000.0 + tim.tv_usec;
    
    


			ipmacc_prompt((char*)"IPMACC: memory allocation btest\n");
acc_create((void*)btest,IDA*sizeof(double));
			

{


    int i;
}
			


    
    
    
    
    
    
    
    
    

    gettimeofday(&tim, NULL);
    end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
    times [k] = (end - start);
  }


}
	




  

  stats(&meantime, &sd, &hm);
  
  printf("%d: CREATETEST time =  %10.3f microseconds +/- %10.3f\n", n, hm, CONF95 * sd);
  
}

  __global__ void __generated_kernel_region_0(int* arg1);
 
  __global__ void __generated_kernel_region_1(int* arg2,int* arg1);
 
  __global__ void __generated_kernel_region_2(int* arg1,int* arg2,int* arg3,int* arg4);
 
  __global__ void __generated_kernel_region_3(int* arg8,int* arg1,int* arg2,int* arg3,int* arg4,int* arg6,int* arg5,int* arg7);
 
  __global__ void __generated_kernel_region_4(int* arg8,int* arg9,int* arg13,int* arg1,int* arg2,int* arg3,int* arg4,int* arg6,int* arg5,int* arg12,int* arg10,int* arg11,int* arg16,int* arg14,int* arg15,int* arg7);
 
void kerneltest()
{
  int n, j, k;
  double start, end;
  double meantime, sd, hm;
  
  n = IDA;
  printf("\n");
  
  

  int *arg1 = (int*)malloc(sizeof(int) * 64);
  int *arg2 = (int*)malloc(sizeof(int) * 64);
  int *arg3 = (int*)malloc(sizeof(int) * 64);
  int *arg4 = (int*)malloc(sizeof(int) * 64);
  int *arg5 = (int*)malloc(sizeof(int) * 64);
  int *arg6 = (int*)malloc(sizeof(int) * 64);
  int *arg7 = (int*)malloc(sizeof(int) * 64);
  int *arg8 = (int*)malloc(sizeof(int) * 64);
  int *arg9 = (int*)malloc(sizeof(int) * 64);
  int *arg10 = (int*)malloc(sizeof(int) * 64);
  int *arg11 = (int*)malloc(sizeof(int) * 64);
  int *arg12 = (int*)malloc(sizeof(int) * 64);
  int *arg13 = (int*)malloc(sizeof(int) * 64);
  int *arg14 = (int*)malloc(sizeof(int) * 64);
  int *arg15 = (int*)malloc(sizeof(int) * 64);
  int *arg16 = (int*)malloc(sizeof(int) * 64);


	ipmacc_prompt((char*)"IPMACC: memory allocation arg1\n");
acc_create((void*)arg1,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg2\n");
acc_create((void*)arg2,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg3\n");
acc_create((void*)arg3,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg4\n");
acc_create((void*)arg4,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg5\n");
acc_create((void*)arg5,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg6\n");
acc_create((void*)arg6,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg7\n");
acc_create((void*)arg7,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg8\n");
acc_create((void*)arg8,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg9\n");
acc_create((void*)arg9,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg10\n");
acc_create((void*)arg10,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg11\n");
acc_create((void*)arg11,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg12\n");
acc_create((void*)arg12,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg13\n");
acc_create((void*)arg13,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg14\n");
acc_create((void*)arg14,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg15\n");
acc_create((void*)arg15,(64+0)*sizeof(int));
ipmacc_prompt((char*)"IPMACC: memory allocation arg16\n");
acc_create((void*)arg16,(64+0)*sizeof(int));
	

{



  {
    
    
		for(k = 0; k <= OUTERREPS; k++)
 {
      gettimeofday(&tim, NULL);
      start = tim.tv_sec * 1000000.0 + tim.tv_usec;



						ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg1\n");
acc_present((void*)arg1);

/* kernel call statement [2, 3]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 0 > gridDim: %d\tblockDim: %d\n",(((abs((int)((1))-0))/(1)))/256+1,256);
__generated_kernel_region_0<<<(((abs((int)((1))-0))/(1)))/256+1,256>>>(
(int*)acc_deviceptr((void*)arg1));
/* kernel call statement*/
			if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();



      gettimeofday(&tim, NULL);
      end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
      times [k] = (end - start);
    }


    stats(&meantime, &sd, &hm);
    printf("%d: Kernel#%d time =  %10.3f microseconds +/- %10.3f\n", n, 1, hm, CONF95 * sd);

    
    
		for(k = 0; k <= OUTERREPS; k++)
 {
      gettimeofday(&tim, NULL);
      start = tim.tv_sec * 1000000.0 + tim.tv_usec;



						ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg1\n");
acc_present((void*)arg1);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg2\n");
acc_present((void*)arg2);

/* kernel call statement [2, 4]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 1 > gridDim: %d\tblockDim: %d\n",(((abs((int)((1))-0))/(1)))/256+1,256);
__generated_kernel_region_1<<<(((abs((int)((1))-0))/(1)))/256+1,256>>>(
(int*)acc_deviceptr((void*)arg2),
(int*)acc_deviceptr((void*)arg1));
/* kernel call statement*/
			if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();



      gettimeofday(&tim, NULL);
      end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
      times [k] = (end - start);
    }


    stats(&meantime, &sd, &hm);
    printf("%d: Kernel#%d time =  %10.3f microseconds +/- %10.3f\n", n, 2, hm, CONF95 * sd);

    
    
		for(k = 0; k <= OUTERREPS; k++)
 {
      gettimeofday(&tim, NULL);
      start = tim.tv_sec * 1000000.0 + tim.tv_usec;



						ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg1\n");
acc_present((void*)arg1);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg2\n");
acc_present((void*)arg2);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg3\n");
acc_present((void*)arg3);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg4\n");
acc_present((void*)arg4);

/* kernel call statement [2, 5]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 2 > gridDim: %d\tblockDim: %d\n",(((abs((int)((1))-0))/(1)))/256+1,256);
__generated_kernel_region_2<<<(((abs((int)((1))-0))/(1)))/256+1,256>>>(
(int*)acc_deviceptr((void*)arg1),
(int*)acc_deviceptr((void*)arg2),
(int*)acc_deviceptr((void*)arg3),
(int*)acc_deviceptr((void*)arg4));
/* kernel call statement*/
			if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();



      gettimeofday(&tim, NULL);
      end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
      times [k] = (end - start);
    }


    stats(&meantime, &sd, &hm);
    printf("%d: Kernel#%d time =  %10.3f microseconds +/- %10.3f\n", n, 4, hm, CONF95 * sd);

    
    
		for(k = 0; k <= OUTERREPS; k++)
 {
      gettimeofday(&tim, NULL);
      start = tim.tv_sec * 1000000.0 + tim.tv_usec;



						ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg1\n");
acc_present((void*)arg1);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg2\n");
acc_present((void*)arg2);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg3\n");
acc_present((void*)arg3);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg4\n");
acc_present((void*)arg4);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg5\n");
acc_present((void*)arg5);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg6\n");
acc_present((void*)arg6);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg7\n");
acc_present((void*)arg7);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg8\n");
acc_present((void*)arg8);

/* kernel call statement [2, 6]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 3 > gridDim: %d\tblockDim: %d\n",(((abs((int)((1))-0))/(1)))/256+1,256);
__generated_kernel_region_3<<<(((abs((int)((1))-0))/(1)))/256+1,256>>>(
(int*)acc_deviceptr((void*)arg8),
(int*)acc_deviceptr((void*)arg1),
(int*)acc_deviceptr((void*)arg2),
(int*)acc_deviceptr((void*)arg3),
(int*)acc_deviceptr((void*)arg4),
(int*)acc_deviceptr((void*)arg6),
(int*)acc_deviceptr((void*)arg5),
(int*)acc_deviceptr((void*)arg7));
/* kernel call statement*/
			if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();



      gettimeofday(&tim, NULL);
      end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
      times [k] = (end - start);
    }


    stats(&meantime, &sd, &hm);
    printf("%d: Kernel#%d time =  %10.3f microseconds +/- %10.3f\n", n, 8, hm, CONF95 * sd);

    
    
		for(k = 0; k <= OUTERREPS; k++)
 {
      gettimeofday(&tim, NULL);
      start = tim.tv_sec * 1000000.0 + tim.tv_usec;



						ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg1\n");
acc_present((void*)arg1);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg2\n");
acc_present((void*)arg2);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg3\n");
acc_present((void*)arg3);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg4\n");
acc_present((void*)arg4);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg5\n");
acc_present((void*)arg5);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg6\n");
acc_present((void*)arg6);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg7\n");
acc_present((void*)arg7);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg8\n");
acc_present((void*)arg8);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg9\n");
acc_present((void*)arg9);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg10\n");
acc_present((void*)arg10);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg11\n");
acc_present((void*)arg11);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg12\n");
acc_present((void*)arg12);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg13\n");
acc_present((void*)arg13);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg14\n");
acc_present((void*)arg14);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg15\n");
acc_present((void*)arg15);
ipmacc_prompt((char*)"IPMACC: memory getting device pointer for arg16\n");
acc_present((void*)arg16);

/* kernel call statement [2, 7]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 4 > gridDim: %d\tblockDim: %d\n",(((abs((int)((1))-0))/(1)))/256+1,256);
__generated_kernel_region_4<<<(((abs((int)((1))-0))/(1)))/256+1,256>>>(
(int*)acc_deviceptr((void*)arg8),
(int*)acc_deviceptr((void*)arg9),
(int*)acc_deviceptr((void*)arg13),
(int*)acc_deviceptr((void*)arg1),
(int*)acc_deviceptr((void*)arg2),
(int*)acc_deviceptr((void*)arg3),
(int*)acc_deviceptr((void*)arg4),
(int*)acc_deviceptr((void*)arg6),
(int*)acc_deviceptr((void*)arg5),
(int*)acc_deviceptr((void*)arg12),
(int*)acc_deviceptr((void*)arg10),
(int*)acc_deviceptr((void*)arg11),
(int*)acc_deviceptr((void*)arg16),
(int*)acc_deviceptr((void*)arg14),
(int*)acc_deviceptr((void*)arg15),
(int*)acc_deviceptr((void*)arg7));
/* kernel call statement*/
			if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();



      gettimeofday(&tim, NULL);
      end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
      times [k] = (end - start);
    }


    stats(&meantime, &sd, &hm);
    printf("%d: Kernel#%d time =  %10.3f microseconds +/- %10.3f\n", n, 16, hm, CONF95 * sd);
  }
}
	


}

void copyintest()
{
  int n, j, k;
  double start, end;
  double meantime, sd, hm;
  
  n = IDA;
  
  
  


	ipmacc_prompt((char*)"IPMACC: memory allocation btest\n");
acc_create((void*)btest,IDA*sizeof(double));
	

{


  
		for(k = 0; k <= OUTERREPS; k++)
 {
    
    gettimeofday(&tim, NULL);
    start = tim.tv_sec * 1000000.0 + tim.tv_usec;
    
    



			ipmacc_prompt((char*)"IPMACC: memory allocation btest\n");
acc_present_or_create((void*)btest,IDA*sizeof(double));
			ipmacc_prompt((char*)"IPMACC: memory copyin btest\n");
acc_pcopyin((void*)btest,IDA*sizeof(double));


{


    int i;
}
			


    
    
    
    
    
    
    

    gettimeofday(&tim, NULL);
    end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
    times [k] = (end - start);
  }


}
	




  

  stats(&meantime, &sd, &hm);
  
  printf("%d: COPYIN time =  %10.3f microseconds +/- %10.3f\n", n, hm, CONF95 * sd);
  
}

void copyouttest()
{
  int n, j, k;
  double start, end;
  double meantime, sd, hm;
  
  n = IDA;
  
  
  


	ipmacc_prompt((char*)"IPMACC: memory allocation btest\n");
acc_create((void*)btest,IDA*sizeof(double));
	

{


  
		for(k = 0; k <= OUTERREPS; k++)
 {
    
    gettimeofday(&tim, NULL);
    start = tim.tv_sec * 1000000.0 + tim.tv_usec;
    
    



			ipmacc_prompt((char*)"IPMACC: memory allocation btest\n");
acc_present_or_create((void*)btest,IDA*sizeof(double));
			

{


    int i;
}
			ipmacc_prompt((char*)"IPMACC: memory copyout btest\n");
acc_copyout_and_keep((void*)btest,IDA*sizeof(double));



    
    
    
    
    
    
    

    gettimeofday(&tim, NULL);
    end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
    times [k] = (end - start);
  }


}
	




  

  stats(&meantime, &sd, &hm);
  
  printf("%d: COPYOUT time =  %10.3f microseconds +/- %10.3f\n", n, hm, CONF95 * sd);

  
}

  __global__ void __generated_kernel_region_5(double* btest,double* result__ipmacc_reductionarray_internal);
 
  __global__ void __generated_kernel_region_6(double* btest,double* result__ipmacc_reductionarray_internal);
 
void reductiontest()
{
  int n, j, k;
  double start, end;
  double meantime, sd, hm;
  double result = 0;
  
  n = IDA;
  printf("\n");
  
  
  

	ipmacc_prompt((char*)"IPMACC: memory allocation btest\n");
acc_create((void*)btest,IDA*sizeof(double));
	ipmacc_prompt((char*)"IPMACC: memory copyin btest\n");
acc_copyin((void*)btest,IDA*sizeof(double));


{


  {
    
    
		for(k = 0; k <= OUTERREPS; k++)
 {
      gettimeofday(&tim, NULL);
      start = tim.tv_sec * 1000000.0 + tim.tv_usec;

      

ipmacc_prompt((char*)"IPMACC: memory allocation result\n");
double* __ipmacc_reduction_array_result=NULL;
if(__ipmacc_reduction_array_result==NULL){
__ipmacc_reduction_array_result=(double*)malloc((((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));
acc_create((void*)__ipmacc_reduction_array_result,(((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));
for(int __ipmacc_initialize_rv=0; __ipmacc_initialize_rv<(((abs((int)((IDA))-0))/(1))/256+1); __ipmacc_initialize_rv++){
__ipmacc_reduction_array_result[__ipmacc_initialize_rv]= 0;
}
acc_pcopyin((void*)__ipmacc_reduction_array_result,(((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));
}

/* kernel call statement [-1, 12]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 5 > gridDim: %d\tblockDim: %d\n",(((abs((int)((IDA))-0))/(1)))/256+1,256);
__generated_kernel_region_5<<<(((abs((int)((IDA))-0))/(1)))/256+1,256>>>(
(double*)acc_deviceptr((void*)btest),
(double*)acc_deviceptr((void*)__ipmacc_reduction_array_result));
/* kernel call statement*/
ipmacc_prompt((char*)"IPMACC: memory copyout result\n");
acc_copyout_and_keep((void*)__ipmacc_reduction_array_result,(((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));

/* second-level reduction on result */
{
int __kernel_reduction_iterator=0;
{
int bound = (((abs((int)((IDA))-0))/(1))/256+1)-1;
for(__kernel_reduction_iterator=bound; __kernel_reduction_iterator>0; __kernel_reduction_iterator-=1){
__ipmacc_reduction_array_result[__kernel_reduction_iterator-1]=__ipmacc_reduction_array_result[__kernel_reduction_iterator-1]+__ipmacc_reduction_array_result[__kernel_reduction_iterator];
}
}
}
result=__ipmacc_reduction_array_result[0];
free(__ipmacc_reduction_array_result);
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();




      gettimeofday(&tim, NULL);
      end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
      times [k] = (end - start);
    }


    stats(&meantime, &sd, &hm);
    printf("%d: REDUCTION(+) time =  %10.3f microseconds +/- %10.3f\n", n, hm, CONF95 * sd);

    
    
		for(k = 0; k <= OUTERREPS; k++)
 {
      gettimeofday(&tim, NULL);
      start = tim.tv_sec * 1000000.0 + tim.tv_usec;

      

ipmacc_prompt((char*)"IPMACC: memory allocation result\n");
double* __ipmacc_reduction_array_result=NULL;
if(__ipmacc_reduction_array_result==NULL){
__ipmacc_reduction_array_result=(double*)malloc((((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));
acc_create((void*)__ipmacc_reduction_array_result,(((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));
for(int __ipmacc_initialize_rv=0; __ipmacc_initialize_rv<(((abs((int)((IDA))-0))/(1))/256+1); __ipmacc_initialize_rv++){
__ipmacc_reduction_array_result[__ipmacc_initialize_rv]= INT_MAX;
}
acc_pcopyin((void*)__ipmacc_reduction_array_result,(((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));
}

/* kernel call statement [-1, 12]*/
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Launching kernel 6 > gridDim: %d\tblockDim: %d\n",(((abs((int)((IDA))-0))/(1)))/256+1,256);
__generated_kernel_region_6<<<(((abs((int)((IDA))-0))/(1)))/256+1,256>>>(
(double*)acc_deviceptr((void*)btest),
(double*)acc_deviceptr((void*)__ipmacc_reduction_array_result));
/* kernel call statement*/
ipmacc_prompt((char*)"IPMACC: memory copyout result\n");
acc_copyout_and_keep((void*)__ipmacc_reduction_array_result,(((abs((int)((IDA))-0))/(1))/256+1)*sizeof(double));

/* second-level reduction on result */
{
int __kernel_reduction_iterator=0;
{
int bound = (((abs((int)((IDA))-0))/(1))/256+1)-1;
for(__kernel_reduction_iterator=bound; __kernel_reduction_iterator>0; __kernel_reduction_iterator-=1){
__ipmacc_reduction_array_result[__kernel_reduction_iterator-1]=(__ipmacc_reduction_array_result[__kernel_reduction_iterator-1]>__ipmacc_reduction_array_result[__kernel_reduction_iterator]?__ipmacc_reduction_array_result[__kernel_reduction_iterator-1]:__ipmacc_reduction_array_result[__kernel_reduction_iterator]);
}
}
}
result=__ipmacc_reduction_array_result[0];
free(__ipmacc_reduction_array_result);
if (getenv("IPMACC_VERBOSE")) printf("IPMACC: Synchronizing the region with host\n");
hipDeviceSynchronize();




      gettimeofday(&tim, NULL);
      end = tim.tv_sec * 1000000.0 + (tim.tv_usec);
      times [k] = (end - start);
    }


    stats(&meantime, &sd, &hm);
    printf("%d: REDUCTION(max) time =  %10.3f microseconds +/- %10.3f\n", n, hm, CONF95 * sd);
  }
}
	


  reftime = meantime;
  refsd = sd;
}

void privatetest()
{
  int j, k;
  int i = 0;
  double start;
  double meantime, sd;
  
  double a;
  

  printf("\n");
  printf("--------------------------------------------------------\n");
  printf("Computing REDUCTION time 1\n");

  for (k = 0; k <= OUTERREPS; k++) {
    start = getclock();
    
    
    for (j = 0; j < innerreps; j++) {
      delay(delaylength, &a);
      i++;
    }
    times [k] = (getclock() - start) * 1.0e6 / (double)innerreps;
  }

  

  printf("PRIVATE time =                           %10.3f microseconds +/- %10.3f\n", meantime, CONF95 * sd);
  printf("PRIVATE overhead =                       %10.3f microseconds +/- %10.3f\n", meantime - reftime, CONF95 * (sd + refsd));

  reftime = meantime;
  refsd = sd;
}

void stats(double *mtp, double *sdp, double *hm)
{
  double meantime, totaltime, sumsq, mintime, maxtime, sd, cutoff;
  double reciprocal, harmonic_mean;
  int i, nr;

  mintime = 1.0e10;
  maxtime = 0.;
  totaltime = 0.;

  for (i = 1; i <= OUTERREPS; i++) {
    mintime = (mintime < times [i]) ? mintime : times [i];
    maxtime = (maxtime > times [i]) ? maxtime : times [i];
    totaltime += times [i];
    reciprocal += 1 / times [i];
  }

  meantime = totaltime / OUTERREPS;
  harmonic_mean = OUTERREPS / reciprocal;
  sumsq = 0;

  for (i = 1; i <= OUTERREPS; i++) {
    sumsq += (times [i] - meantime) * (times [i] - meantime);
  }
  sd = sqrt(sumsq / (OUTERREPS - 1));

  cutoff = 3.0 * sd;

  nr = 0;

  for (i = 1; i <= OUTERREPS; i++) {
    if (fabs(times [i] - meantime) > cutoff) {
      nr++;
    }
  }

  
  
  
  

  *mtp = meantime;
  *sdp = sd;
  *hm = harmonic_mean;
}


 __global__ void __generated_kernel_region_0(int* arg1){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
{
{
{
int i=0+(__kernel_getuid);
if( i < 1)
{
        arg1 [i] = i;
      }

}
}
}
}
 __global__ void __generated_kernel_region_1(int* arg2,int* arg1){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
{
{
{
int i=0+(__kernel_getuid);
if( i < 1)
{
        arg1 [i] = i;
        arg2 [i] = i;
      }

}
}
}
}
 __global__ void __generated_kernel_region_2(int* arg1,int* arg2,int* arg3,int* arg4){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
{
{
{
int i=0+(__kernel_getuid);
if( i < 1)
{
        arg1 [i] = i;
        arg2 [i] = i;
        arg3 [i] = i;
        arg4 [i] = i;
      }

}
}
}
}
 __global__ void __generated_kernel_region_3(int* arg8,int* arg1,int* arg2,int* arg3,int* arg4,int* arg6,int* arg5,int* arg7){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
{
{
{
int i=0+(__kernel_getuid);
if( i < 1)
{
        arg1 [i] = i;
        arg2 [i] = i;
        arg3 [i] = i;
        arg4 [i] = i;
        arg5 [i] = i;
        arg6 [i] = i;
        arg7 [i] = i;
        arg8 [i] = i;
      }

}
}
}
}
 __global__ void __generated_kernel_region_4(int* arg8,int* arg9,int* arg13,int* arg1,int* arg2,int* arg3,int* arg4,int* arg6,int* arg5,int* arg12,int* arg10,int* arg11,int* arg16,int* arg14,int* arg15,int* arg7){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
{
{
{
int i=0+(__kernel_getuid);
if( i < 1)
{
        arg1 [i] = i;
        arg2 [i] = i;
        arg3 [i] = i;
        arg4 [i] = i;
        arg5 [i] = i;
        arg6 [i] = i;
        arg7 [i] = i;
        arg8 [i] = i;
        arg9 [i] = i;
        arg10 [i] = i;
        arg11 [i] = i;
        arg12 [i] = i;
        arg13 [i] = i;
        arg14 [i] = i;
        arg15 [i] = i;
        arg16 [i] = i;
      }

}
}
}
}
 __global__ void __generated_kernel_region_5(double* btest,double* result__ipmacc_reductionarray_internal){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
__shared__ double __kernel_reduction_shmem_double[256];
int j;
int __kernel_reduction_iterator=0;
double result;{
{
{
 j=0+(__kernel_getuid);
if( j < IDA)
{ //opened for private and reduction
/*private:+:result*/
{ //start of reduction region for result 
double result=0;

{
        double x = btest [j];
        result += x;
      }
/*reduction:+:result*/

/* reduction on result */
__syncthreads();
__kernel_reduction_shmem_double[threadIdx.x]=result;
__syncthreads();
for(__kernel_reduction_iterator=blockDim.x/2;__kernel_reduction_iterator>0; __kernel_reduction_iterator>>=1) {
if(threadIdx.x<__kernel_reduction_iterator){
__kernel_reduction_shmem_double[threadIdx.x]=__kernel_reduction_shmem_double[threadIdx.x]+__kernel_reduction_shmem_double[threadIdx.x+__kernel_reduction_iterator];
}
__syncthreads();
}
}// the end of result scope
if(threadIdx.x==0){
result__ipmacc_reductionarray_internal[blockIdx.x]=__kernel_reduction_shmem_double[0];
}

} // closed for reduction-end

}
}
}
}
 __global__ void __generated_kernel_region_6(double* btest,double* result__ipmacc_reductionarray_internal){
int __kernel_getuid=threadIdx.x+blockIdx.x*blockDim.x;
__shared__ double __kernel_reduction_shmem_double[256];
int j;
int __kernel_reduction_iterator=0;
double result;{
{
{
 j=0+(__kernel_getuid);
if( j < IDA)
{ //opened for private and reduction
/*private:max:result*/
{ //start of reduction region for result 
double result=0;

{
        double x = btest [j];
        result += x;
      }
/*reduction:max:result*/

/* reduction on result */
__syncthreads();
__kernel_reduction_shmem_double[threadIdx.x]=result;
__syncthreads();
for(__kernel_reduction_iterator=blockDim.x/2;__kernel_reduction_iterator>0; __kernel_reduction_iterator>>=1) {
if(threadIdx.x<__kernel_reduction_iterator){
__kernel_reduction_shmem_double[threadIdx.x]=(__kernel_reduction_shmem_double[threadIdx.x]>__kernel_reduction_shmem_double[threadIdx.x+__kernel_reduction_iterator]?__kernel_reduction_shmem_double[threadIdx.x]:__kernel_reduction_shmem_double[threadIdx.x+__kernel_reduction_iterator]);
}
__syncthreads();
}
}// the end of result scope
if(threadIdx.x==0){
result__ipmacc_reductionarray_internal[blockIdx.x]=__kernel_reduction_shmem_double[0];
}

} // closed for reduction-end

}
}
}
}

